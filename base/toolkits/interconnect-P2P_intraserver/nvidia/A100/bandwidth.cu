// Copyright (c) 2024 BAAI. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License")

#include <stdio.h>
#include <hip/hip_runtime.h>

#define SIZE (1024ULL * 1024ULL * 1024ULL * sizeof(float))
#define WARMUP_ITERATIONS 100
#define ITERATIONS 20000

void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main() {
    float *d_src, *d_dst;
    hipEvent_t start, end;
    float elapsed_time;
    int gpu_n;
    checkCudaError(hipGetDeviceCount(&gpu_n), "hipGetDeviceCount");
    printf("[FlagPerf Info]CUDA-capable device count: %i\n", gpu_n);
    if (gpu_n < 2) {
        fprintf(stderr, "Two or more GPUs with Peer-to-Peer access capability are required for inferconnect-P2P_intraserver-bandwidth test\n");
        exit(EXIT_FAILURE);
    }
    int can_access_peer;
    int p2pCapableGPUs[2];  // We take only 1 pair of P2P capable GPUs
    p2pCapableGPUs[0] = p2pCapableGPUs[1] = -1;

    // Show all the combinations of supported P2P GPUs
    for (int i = 0; i < gpu_n; i++) {
        for (int j = 0; j < gpu_n; j++) {
            if (i == j) {
                continue;
            }
            checkCudaError(hipDeviceCanAccessPeer(&can_access_peer, i, j), "hipDeviceCanAccessPeer");
            printf("[FlagPerf Info]> Peer access from (GPU%d) -> (GPU%d) : %s\n",
                    i, j, can_access_peer ? "Yes" : "No");
            if (can_access_peer && p2pCapableGPUs[0] == -1) {
                p2pCapableGPUs[0] = i;
                p2pCapableGPUs[1] = j;
            }
        }
    }
    if (p2pCapableGPUs[0] == -1 || p2pCapableGPUs[1] == -1) {
        printf(
            "[FlagPerf Info]Two or more GPUs with Peer-to-Peer access capability are required for inferconnect-P2P_intraserver-bandwidth test\n");
        printf(
            "[FlagPerf Info]Peer to Peer access is not available amongst GPUs in the system, "
            "waiving test.\n");
        return 0;
    }
    int gpuid[2];
    gpuid[0] = p2pCapableGPUs[0];
    gpuid[1] = p2pCapableGPUs[1];
    printf("[FlagPerf Info]Enabling peer access between GPU%d and GPU%d...\n", gpuid[0],
            gpuid[1]);
    printf("Allocating buffers (%iGB on GPU%d, GPU%d and CPU Host)...\n",
         int(SIZE / 1024 / 1024 / 1024), gpuid[0], gpuid[1]);

    checkCudaError(hipSetDevice(gpuid[0]), "hipSetDevice");
    checkCudaError(hipDeviceEnablePeerAccess(gpuid[1], 0), "hipDeviceEnablePeerAccess");
    checkCudaError(hipSetDevice(gpuid[1]), "hipSetDevice");
    checkCudaError(hipDeviceEnablePeerAccess(gpuid[0], 0), "hipDeviceEnablePeerAccess");

    checkCudaError(hipSetDevice(gpuid[0]), "hipSetDevice");
    checkCudaError(hipMalloc(&d_src, SIZE), "hipMalloc");
    checkCudaError(hipSetDevice(gpuid[1]), "hipSetDevice");
    checkCudaError(hipMalloc(&d_dst, SIZE), "hipMalloc");
    
    checkCudaError(hipEventCreate(&start), "hipEventCreate");
    checkCudaError(hipEventCreate(&end), "hipEventCreate");


    for (int i = 0; i < WARMUP_ITERATIONS; ++i) {
        if (i % 2 == 0) {
            checkCudaError(hipMemcpy(d_dst, d_src, SIZE, hipMemcpyDefault), "hipMemcpy");
        } else {
            checkCudaError(hipMemcpy(d_src, d_dst, SIZE, hipMemcpyDefault), "hipMemcpy");
        }
    }


    checkCudaError(hipEventRecord(start, 0), "hipEventRecord");

    for (int i = 0; i < ITERATIONS; ++i) {
        if (i % 2 == 0) {
            checkCudaError(hipMemcpy(d_dst, d_src, SIZE, hipMemcpyDefault), "hipMemcpy");
        } else {
            checkCudaError(hipMemcpy(d_src, d_dst, SIZE, hipMemcpyDefault), "hipMemcpy");
        } 
    }

    checkCudaError(hipEventRecord(end, 0), "hipEventRecord");
    checkCudaError(hipEventSynchronize(end), "hipEventSynchronize");

    checkCudaError(hipEventElapsedTime(&elapsed_time, start, end), "hipEventElapsedTime");

    double bandwidth = 2.0 * SIZE * ITERATIONS / (elapsed_time / 1000.0);

    printf("[FlagPerf Result]inferconnect-P2P_intraserver-bandwidth=%.2fGiB/s\n", bandwidth / (1024.0 * 1024.0 * 1024.0));
    printf("[FlagPerf Result]inferconnect-P2P_intraserver-bandwidth=%.2fGB/s\n", bandwidth / (1000.0 * 1000.0 * 1000.0));


    checkCudaError(hipSetDevice(gpuid[0]), "hipSetDevice");
    checkCudaError(hipDeviceDisablePeerAccess(gpuid[1]), "hipDeviceDisablePeerAccess");
    checkCudaError(hipSetDevice(gpuid[1]), "hipSetDevice");
    checkCudaError(hipDeviceDisablePeerAccess(gpuid[0]), "hipDeviceDisablePeerAccess");

    checkCudaError(hipFree(d_src), "hipFree");
    checkCudaError(hipFree(d_dst), "hipFree");
    checkCudaError(hipEventDestroy(start), "hipEventDestroy");
    checkCudaError(hipEventDestroy(end), "hipEventDestroy");

    return 0;
}
