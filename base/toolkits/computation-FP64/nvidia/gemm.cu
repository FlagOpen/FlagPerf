#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>

constexpr int M = 8192;
constexpr int N = 8192;
constexpr int K = 8192;

struct PrecisionConfig {
    hipDataType cudaType;
    hipblasComputeType_t cublasType;
    int bytesPerElement;
    const char* name;
    int NUM_ITERATIONS ;
    int WARMUP_ITERATIONS = 10;
};

void test(const PrecisionConfig& config) {
    double  *d_A, *d_B, *d_C;

    hipMallocManaged(&d_A, M * K * config.bytesPerElement);
    hipMallocManaged(&d_B, K * N * config.bytesPerElement);
    if (config.cudaType == HIP_R_8I) {
        hipMallocManaged(&d_C, M * N * sizeof(float));
    } else {
        hipMallocManaged(&d_C, M * N * config.bytesPerElement);
    }

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    double alpha = 1.0;
    double beta = 0.0;

    for (int i = 0; i < config.WARMUP_ITERATIONS; ++i) {
        if (config.cudaType == HIP_R_8I) {
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                         M, N, K, &alpha,
                         d_A, config.cudaType, M,
                         d_B, config.cudaType, K,
                         &beta,
                         d_C, HIP_R_32I, M,
                         config.cublasType, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        } else {
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                         M, N, K, &alpha,
                         d_A, config.cudaType, M,
                         d_B, config.cudaType, K,
                         &beta,
                         d_C, config.cudaType, M,
                         config.cublasType, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        }
    }

    hipError_t syncError = hipDeviceSynchronize();
    auto start = std::chrono::high_resolution_clock::now();

    if (syncError != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(syncError) << std::endl;
    }

    for (int i = 0; i < config.NUM_ITERATIONS; ++i) {
        if (config.cudaType == HIP_R_8I) {
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                         M, N, K, &alpha,
                         d_A, config.cudaType, M,
                         d_B, config.cudaType, K,
                         &beta,
                         d_C, HIP_R_32I, M,
                         config.cublasType, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        } else {
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                         M, N, K, &alpha,
                         d_A, config.cudaType, M,
                         d_B, config.cudaType, K,
                         &beta,
                         d_C, config.cudaType, M,
                         config.cublasType, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        }
    }
    syncError = hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    if (syncError != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(syncError) << std::endl;
    }
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    std::cout << "Average " << config.name << " Single Op Duration: " << duration.count() / config.NUM_ITERATIONS << " us" << std::endl;

    double time_second = duration.count() / 1.0e6;
    double flops = 2.0 * M * N * K * config.NUM_ITERATIONS;
    double FLOPS = flops / time_second;
    double TFLOPS = FLOPS / 1.0e12;

    std::cout << "[FlagPerf Result]" << "computation-FP64=" << TFLOPS << "TFLOPS" << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipblasDestroy(handle);
}

int main() {
    PrecisionConfig fp64 = {
        HIP_R_64F,
        HIPBLAS_COMPUTE_64F,
        8,
        "FP64",
        10000,
        10
    };

    test(fp64);

    return 0;
}

