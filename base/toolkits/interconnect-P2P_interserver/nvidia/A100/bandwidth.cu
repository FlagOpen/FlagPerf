// Copyright (c) 2024 BAAI. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License")
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <nccl.h>
#include <mpi.h>

#define SIZE (1024ULL * 1024ULL * 1024ULL * sizeof(float))
#define WARMUP_ITERATIONS 1000
#define ITERATIONS 5000

void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void checkNcclError(ncclResult_t result, const char *msg) {
    if (result != ncclSuccess) {
        fprintf(stderr, "NCCL Error: %s: %s\n", msg, ncclGetErrorString(result));
        exit(EXIT_FAILURE);
    }
}

void checkMPIError(int result, const char *msg) {
    if (result != MPI_SUCCESS) {
        char error_string[MPI_MAX_ERROR_STRING];
        int length;
        MPI_Error_string(result, error_string, &length);
        fprintf(stderr, "MPI Error: %s: %s\n", msg, error_string);
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char **argv) {
    float *d_tensor;
    hipEvent_t start, end;
    float elapsed_time;

    checkMPIError(MPI_Init(&argc, &argv), "MPI_Init");
    int rank, nranks;
    checkMPIError(MPI_Comm_rank(MPI_COMM_WORLD, &rank), "MPI_Comm_rank");
    checkMPIError(MPI_Comm_size(MPI_COMM_WORLD, &nranks), "MPI_Comm_size");
    checkCudaError(hipSetDevice(0), "hipSetDevice");

    ncclComm_t comm;
    hipStream_t stream;

    ncclUniqueId id;
    if (rank == 0) {
        checkNcclError(ncclGetUniqueId(&id), "ncclGetUniqueId");
    }
    MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);

    checkNcclError(ncclCommInitRank(&comm, nranks, id, rank), "ncclCommInitRank");
    checkCudaError(hipStreamCreate(&stream), "hipStreamCreate");
    
    checkCudaError(hipMalloc(&d_tensor, SIZE), "hipMalloc");

    checkCudaError(hipEventCreate(&start), "hipEventCreate");
    checkCudaError(hipEventCreate(&end), "hipEventCreate");

    checkNcclError(ncclGroupStart(), "ncclGroupStart");
    for (int i = 0; i < WARMUP_ITERATIONS; ++i) {
        if (rank == 0) {
            checkNcclError(ncclSend(d_tensor, SIZE / sizeof(float), ncclFloat, 1, comm, stream), "ncclSend");
        }
        else if (rank == 1){
            checkNcclError(ncclRecv(d_tensor, SIZE / sizeof(float), ncclFloat, 0, comm, stream), "ncclRecv");
        }
    }
    checkNcclError(ncclGroupEnd(), "ncclGroupEnd");
    checkCudaError(hipStreamSynchronize(stream), "hipStreamSynchronize");
    checkMPIError(MPI_Barrier(MPI_COMM_WORLD), "MPI_Barrier");

    checkCudaError(hipEventRecord(start), "hipEventRecord");
    checkNcclError(ncclGroupStart(), "ncclGroupStart");
    for (int i = 0; i < ITERATIONS; ++i) {
        if (rank == 0) {
            checkNcclError(ncclSend(d_tensor, SIZE / sizeof(float), ncclFloat, 1, comm, stream), "ncclSend");
        }
        else if (rank == 1){
            checkNcclError(ncclRecv(d_tensor, SIZE / sizeof(float), ncclFloat, 0, comm, stream), "ncclRecv");
        }
    }
    checkNcclError(ncclGroupEnd(), "ncclGroupEnd");
    checkCudaError(hipStreamSynchronize(stream), "hipStreamSynchronize");
    checkMPIError(MPI_Barrier(MPI_COMM_WORLD), "MPI_Barrier");
    checkCudaError(hipEventRecord(end), "hipEventRecord");
    checkCudaError(hipEventSynchronize(end), "hipEventSynchronize");
    checkCudaError(hipEventElapsedTime(&elapsed_time, start, end), "hipEventElapsedTime");

    double bandwidth = SIZE * ITERATIONS / (elapsed_time / 1000.0);
    printf("[FlagPerf Result]interconnect-MPI_intraserver-bandwidth=%.2fGiB/s\n", bandwidth / (1024.0 * 1024.0 * 1024.0));
    printf("[FlagPerf Result]interconnect-MPI_intraserver-bandwidth=%.2fGB/s\n", bandwidth / (1000.0 * 1000.0 * 1000.0));

    checkCudaError(hipEventDestroy(start), "hipEventDestroy");
    checkCudaError(hipEventDestroy(end), "hipEventDestroy");
    checkCudaError(hipFree(d_tensor), "hipFree");
    checkNcclError(ncclCommDestroy(comm), "ncclCommDestroy");
    checkCudaError(hipStreamDestroy(stream), "hipStreamDestroy");
    checkMPIError(MPI_Finalize(), "MPI_Finalize");
    return 0;
}
