#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <hip/hip_bf16.h>

constexpr int M = 8192;
constexpr int N = 8192;
constexpr int K = 8192;

struct PrecisionConfig {
    hipDataType cudaType;
    hipblasComputeType_t cublasType;
    int bytesPerElement;
    const char* name;
    int NUM_ITERATIONS ;
    int WARMUP_ITERATIONS = 10;
};

void test(const PrecisionConfig& config) {
    __hip_bfloat16  *d_A, *d_B, *d_C;

    hipMallocManaged(&d_A, M * K * config.bytesPerElement);
    hipMallocManaged(&d_B, K * N * config.bytesPerElement);
    if (config.cudaType == HIP_R_8I) {
        hipMallocManaged(&d_C, M * N * sizeof(float));
    } else {
        hipMallocManaged(&d_C, M * N * config.bytesPerElement);
    }

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    __hip_bfloat16 alpha = __float2bfloat16(1.0f);
    __hip_bfloat16 beta = __float2bfloat16(0.0f);

    for (int i = 0; i < config.WARMUP_ITERATIONS; ++i) {
        if (config.cudaType == HIP_R_8I) {
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                         M, N, K, &alpha,
                         d_A, config.cudaType, M,
                         d_B, config.cudaType, K,
                         &beta,
                         d_C, HIP_R_32I, M,
                         config.cublasType, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        } else {
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                         M, N, K, &alpha,
                         d_A, config.cudaType, M,
                         d_B, config.cudaType, K,
                         &beta,
                         d_C, config.cudaType, M,
                         config.cublasType, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        }
    }

    hipError_t syncError = hipDeviceSynchronize();
    auto start = std::chrono::high_resolution_clock::now();

    if (syncError != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(syncError) << std::endl;
    }

    for (int i = 0; i < config.NUM_ITERATIONS; ++i) {
        if (config.cudaType == HIP_R_8I) {
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                         M, N, K, &alpha,
                         d_A, config.cudaType, M,
                         d_B, config.cudaType, K,
                         &beta,
                         d_C, HIP_R_32I, M,
                         config.cublasType, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        } else {
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                         M, N, K, &alpha,
                         d_A, config.cudaType, M,
                         d_B, config.cudaType, K,
                         &beta,
                         d_C, config.cudaType, M,
                         config.cublasType, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        }
    }
    syncError = hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    if (syncError != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(syncError) << std::endl;
    }
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    std::cout << "Average " << config.name << " Single Op Duration: " << duration.count() / config.NUM_ITERATIONS << " us" << std::endl;

    double time_second = duration.count() / 1.0e6;
    double flops = 2.0 * M * N * K * config.NUM_ITERATIONS;
    double FLOPS = flops / time_second;
    double TFLOPS = FLOPS / 1.0e12;

    std::cout << "[FlagPerf Result]" << "computation-BF16=" << TFLOPS << "TFLOPS" << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipblasDestroy(handle);
}

int main() {
    PrecisionConfig bf16 = {
        HIP_R_16BF,
        HIPBLAS_COMPUTE_32F,
        2,
        "BF16",
        50000,
        10
    };

    test(bf16);

    return 0;
}

