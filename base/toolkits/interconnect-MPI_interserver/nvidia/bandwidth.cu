#include <stdio.h>
#include <hip/hip_runtime.h>
#include <nccl.h>
#include <mpi.h>
#include <vector>
#include <iostream>

#define GB (1024ULL * 1024ULL * 1024ULL)
#define SIZE (4ULL * GB)
#define WARMUP_ITERATIONS 200
#define ITERATIONS 2000

void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void checkNcclError(ncclResult_t result, const char *msg) {
    if (result != ncclSuccess) {
        fprintf(stderr, "NCCL Error: %s: %s\n", msg, ncclGetErrorString(result));
        exit(EXIT_FAILURE);
    }
}

void checkMPIError(int result, const char *msg) {
    if (result != MPI_SUCCESS) {
        fprintf(stderr, "MPI Error: %s\n", msg);
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char *argv[]) {
    checkMPIError(MPI_Init(&argc, &argv), "MPI_Init");

    int rank, size;
    checkMPIError(MPI_Comm_rank(MPI_COMM_WORLD, &rank), "MPI_Comm_rank");
    checkMPIError(MPI_Comm_size(MPI_COMM_WORLD, &size), "MPI_Comm_size");

    int num_gpus_per_node = 8;
    int total_gpus = size;
    int gpu_id = rank % num_gpus_per_node;

    hipEvent_t start, end;
    float elapsed_time;
    float* d_src;
    float* d_dst;
    ncclComm_t comm;
    hipStream_t stream;

    checkCudaError(hipSetDevice(gpu_id), "hipSetDevice");
    checkCudaError(hipMalloc(&d_src, SIZE), "hipMalloc");
    checkCudaError(hipMalloc(&d_dst, SIZE), "hipMalloc");
    checkCudaError(hipMemset(d_src, 1.0f, SIZE), "hipMemset");
    checkCudaError(hipStreamCreate(&stream), "hipStreamCreate");

    ncclUniqueId id;
    if (rank == 0) checkNcclError(ncclGetUniqueId(&id), "ncclGetUniqueId");
    checkMPIError(MPI_Bcast(&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD), "MPI_Bcast");
    checkNcclError(ncclCommInitRank(&comm, total_gpus, id, rank), "ncclCommInitRank");
    checkCudaError(hipEventCreate(&start), "hipEventCreate");
    checkCudaError(hipEventCreate(&end), "hipEventCreate");

    for (int i = 0; i < WARMUP_ITERATIONS; ++i) {
        checkNcclError(ncclAllReduce((const void*)d_src, (void*)d_dst, SIZE / sizeof(float), ncclFloat, ncclSum, comm, stream), "ncclAllReduce");
        checkCudaError(hipStreamSynchronize(stream), "hipStreamSynchronize");
    }
    checkMPIError(MPI_Barrier(MPI_COMM_WORLD), "MPI_Barrier");
    checkCudaError(hipEventRecord(start), "hipEventRecord");

    for (int i = 0; i < ITERATIONS; ++i) {
        checkNcclError(ncclAllReduce((const void*)d_src, (void*)d_dst, SIZE / sizeof(float), ncclFloat, ncclSum, comm, stream), "ncclAllReduce");
        checkCudaError(hipStreamSynchronize(stream), "hipStreamSynchronize");
    }
    checkMPIError(MPI_Barrier(MPI_COMM_WORLD), "MPI_Barrier");
    checkCudaError(hipEventRecord(end), "hipEventRecord"); 
    checkCudaError(hipEventSynchronize(end), "hipEventSynchronize");
    checkCudaError(hipEventElapsedTime(&elapsed_time, start, end), "hipEventElapsedTime");
    double algbw = SIZE * ITERATIONS / (elapsed_time / 1000.0);
    double bandwidth = algbw * (2.0 * (total_gpus - 1) / total_gpus);
    if (rank == 0) {
        printf("[FlagPerf Result]interconnect-MPI_interserver-bandwidth=%.2fGiB/s\n", bandwidth / (1024.0 * 1024.0 * 1024.0));
        printf("[FlagPerf Result]interconnect-MPI_interserver-bandwidth=%.2fGB/s\n", bandwidth / (1000.0 * 1000.0 * 1000.0));
    }
    checkCudaError(hipFree(d_src), "hipFree");
    checkCudaError(hipFree(d_dst), "hipFree");
    checkNcclError(ncclCommDestroy(comm), "ncclCommDestroy");
    checkCudaError(hipEventDestroy(start), "hipEventDestroy");
    checkCudaError(hipEventDestroy(end), "hipEventDestroy");
    checkMPIError(MPI_Finalize(), "MPI_Finalize");
    return 0;
}
