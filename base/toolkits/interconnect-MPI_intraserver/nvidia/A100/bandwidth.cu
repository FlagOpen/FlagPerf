// Copyright (c) 2024 BAAI. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License")
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <nccl.h>
#include <vector>
#include <iostream>


#define GB (1024ULL * 1024ULL * 1024ULL)
#define SIZE (4ULL * GB)
#define WARMUP_ITERATIONS 100
#define ITERATIONS 10000

void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void checkNcclError(ncclResult_t result, const char *msg) {
    if (result != ncclSuccess) {
        fprintf(stderr, "NCCL Error: %s: %s\n", msg, ncclGetErrorString(result));
        exit(EXIT_FAILURE);
    }
}

int main() {
    int num_gpus = 8;
    int devs[num_gpus] = {0, 1, 2, 3, 4, 5, 6, 7};

    hipEvent_t start, end;
    float elapsed_time;
    std::vector<float*> d_src(num_gpus);
    std::vector<float*> d_dst(num_gpus);
    std::vector<ncclComm_t> comms(num_gpus);
    std::vector<hipStream_t> streams(num_gpus);

    for (int i = 0; i < num_gpus; ++i) {
        checkCudaError(hipSetDevice(devs[i]), "hipSetDevice");
        checkCudaError(hipMalloc(&d_src[i], SIZE), "hipMalloc");
        checkCudaError(hipMalloc(&d_dst[i], SIZE), "hipMalloc");
        checkCudaError(hipMemset(d_src[i], 1.0f, SIZE), "hipMemset");
        checkCudaError(hipStreamCreate(&streams[i]), "hipStreamCreate");
    }
    checkNcclError(ncclCommInitAll(comms.data(), num_gpus, devs), "ncclCommInitAll");

    checkCudaError(hipEventCreate(&start), "hipEventCreate");
    checkCudaError(hipEventCreate(&end), "hipEventCreate");
    for (int i = 0; i < WARMUP_ITERATIONS; ++i) {
        checkNcclError(ncclGroupStart(), "ncclGroupStart");
        for (int j = 0; j < num_gpus; ++j) {
            checkNcclError(ncclAllReduce((const void*)d_src[j], (void*)d_dst[j], SIZE / sizeof(float), ncclFloat, ncclSum, comms[j], streams[j]), "ncclAllReduce");
        }
        checkNcclError(ncclGroupEnd(), "ncclGroupEnd");
        for (int j = 0; j < num_gpus; ++j){
            checkCudaError(hipStreamSynchronize(streams[j]), "hipStreamSynchronize");
        } 
    }

    checkCudaError(hipEventRecord(start), "hipEventRecord");

    for (int i = 0; i < ITERATIONS; ++i) {
        checkNcclError(ncclGroupStart(), "ncclGroupStart");
        for (int j = 0; j < num_gpus; ++j) {
            checkNcclError(ncclAllReduce((const void*)d_src[j], (void*)d_dst[j], SIZE / sizeof(float), ncclFloat, ncclSum, comms[j], streams[j]), "ncclAllReduce");
        }
        checkNcclError(ncclGroupEnd(), "ncclGroupEnd");
        for (int j = 0; j < num_gpus; ++j){
            checkCudaError(hipStreamSynchronize(streams[j]), "hipStreamSynchronize");
        }
    }
    checkCudaError(hipEventRecord(end), "hipEventRecord"); 
    checkCudaError(hipEventSynchronize(end), "hipEventSynchronize");
    checkCudaError(hipEventElapsedTime(&elapsed_time, start, end), "hipEventElapsedTime");

    /*
        algbw = S/t
    Considering that each rank has a bandwidth to the outside world of B, the time to perform an allReduce operation of S elements is at best :
        t = (S*2*(n-1)) / (n*B)
    Indeed, we have S elements, 2*(n-1) operations per element, and n links of bandwidth B to perform them. Reordering the equation, we find that
        t = (S/B) * (2*(n-1)/n)
    Therefore, to get an AllReduce bandwidth measurement which we can compare to the hardware peak bandwidth, we compute :
        B = S/t * (2*(n-1)/n) = algbw * (2*(n-1)/n)
    More details can be found in https://github.com/NVIDIA/nccl-tests/blob/master/doc/PERFORMANCE.md
    NVIDIA specifies the 600GBps for intra-server connect as a bidirectional bandwidth, 
    meaning each node can simultaneously upload and download at 300GBps. 
    To better reflect the ratio of the tested value to the specified value and 
    to align with common understanding of NVIDIA's product capabilities, 
    we have multiplied the bandwidth result here by two.
    */
    double algbw = SIZE * ITERATIONS / (elapsed_time / 1000.0);
    double bandwidth = algbw * (2.0 * (num_gpus-1) / num_gpus);
    bandwidth = bandwidth * 2.0;

    printf("[FlagPerf Result]interconnect-MPI_intraserver-bandwidth=%.2fGiB/s\n", bandwidth / (1024.0 * 1024.0 * 1024.0));
    printf("[FlagPerf Result]interconnect-MPI_intraserver-bandwidth=%.2fGB/s\n", bandwidth / (1000.0 * 1000.0 * 1000.0));

    for (int i = 0; i < num_gpus; ++i) {
        checkCudaError(hipFree(d_src[i]), "hipFree");
        checkCudaError(hipFree(d_dst[i]), "hipFree");
        checkNcclError(ncclCommDestroy(comms[i]), "ncclCommDestroy");
    }
    checkCudaError(hipEventDestroy(start), "hipEventDestroy");
    checkCudaError(hipEventDestroy(end), "hipEventDestroy");
    return 0;
}
