#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>


constexpr int M = 8192;
constexpr int N = 8192;
constexpr int K = 8192;

struct PrecisionConfig {
    hipDataType cudaType;
    hipblasComputeType_t cublasType;
    int bytesPerElement;
    const char* name;
    int NUM_ITERATIONS ;
    int WARMUP_ITERATIONS = 10;
};

void test(const PrecisionConfig& config) {
    void  *d_A, *d_B, *d_C;

    hipMallocManaged(&d_A, M * K * config.bytesPerElement);
    hipMallocManaged(&d_B, K * N * config.bytesPerElement);
    if (config.cudaType == HIP_R_8I) {
        hipMallocManaged(&d_C, M * N * sizeof(float));
    } else {
        hipMallocManaged(&d_C, M * N * config.bytesPerElement);
    }

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1.0;
    float beta = 0.0;

    for (int i = 0; i < config.WARMUP_ITERATIONS; ++i) {
        if (config.cudaType == HIP_R_8I) {
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                         M, N, K, &alpha,
                         d_A, config.cudaType, M,
                         d_B, config.cudaType, K,
                         &beta,
                         d_C, HIP_R_32I, M,
                         config.cublasType, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        } else {
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                         M, N, K, &alpha,
                         d_A, config.cudaType, M,
                         d_B, config.cudaType, K,
                         &beta,
                         d_C, config.cudaType, M,
                         config.cublasType, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        }
    }

    hipError_t syncError = hipDeviceSynchronize();
    auto start = std::chrono::high_resolution_clock::now();

    if (syncError != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(syncError) << std::endl;
    }

    for (int i = 0; i < config.NUM_ITERATIONS; ++i) {
        if (config.cudaType == HIP_R_8I) {
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                         M, N, K, &alpha,
                         d_A, config.cudaType, M,
                         d_B, config.cudaType, K,
                         &beta,
                         d_C, HIP_R_32I, M,
                         config.cublasType, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        } else {
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                         M, N, K, &alpha,
                         d_A, config.cudaType, M,
                         d_B, config.cudaType, K,
                         &beta,
                         d_C, config.cudaType, M,
                         config.cublasType, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        }
    }
    syncError = hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    if (syncError != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(syncError) << std::endl;
    }
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    std::cout << "Average " << config.name << " Single Op Duration: " << duration.count() / config.NUM_ITERATIONS << " us" << std::endl;

    double time_second = duration.count() / 1.0e6;
    double ops = 2.0 * M * N * K * config.NUM_ITERATIONS;
    double OPS = ops / time_second;
    double TOPS = OPS / 1.0e12;

    std::cout << "[FlagPerf Result]" << "computation-INT8=" << TOPS << "TOPS" << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipblasDestroy(handle);
}

int main() {
    PrecisionConfig int8 = {
        HIP_R_8I,
        HIPBLAS_COMPUTE_32I,
        1,
        "INT8",
        100000,
        10
    };

    test(int8);

    return 0;
}

